#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <cmath>
#include <string>
#include <hip/hip_runtime.h>
#include <pthread.h>
#include <hip/hip_runtime_api.h>
#include <unistd.h>
#include <sched.h>
#include <sys/mman.h>
#include <sys/wait.h>
#include <linux/version.h>

using namespace std;

#define gpuDriverErrorCheck(ans) {gpuAssert((ans),__FILE__,__LINE__); }
inline void gpuAssert(hipError_t code,  const char *file, int line)
{
    if (code != hipSuccess )
    {
        fprintf(stderr, "GPU assert: %d %s %d\n",
            code, file, line);
    }
}

__global__ void checkDeviceResult(int * ptr, int total_elem, int expect_val) {
    int i = 0;
    while (i < total_elem) {
        if (ptr[i] == expect_val) {
            i+=1;
        } else {
            printf("Error: value incorrect\n");
            break;
        }
    }
}

int main()
{
    const int total_elem = 2<<10;
    const int buf_size = total_elem * sizeof(int);

    //using cuda driver api
    //initialize cuda driver
    gpuDriverErrorCheck(hipInit(0)); 
    int count = 0;
    gpuDriverErrorCheck(hipGetDeviceCount(&count));
    hipDevice_t device;
    gpuDriverErrorCheck(hipDeviceGet(&device, 0));
    hipCtx_t context;
    gpuDriverErrorCheck(hipCtxCreate(&context, 0, device));

    //Initialize value in memory
    hipDeviceptr_t tmp;
    gpuDriverErrorCheck(hipMalloc(&tmp, buf_size));

    hipDeviceptr_t h_mem;
    int* h_mem_ptr;
    gpuDriverErrorCheck(hipHostAlloc((void **)&h_mem_ptr, buf_size,  hipHostMallocMapped)); 
    for(int i = 0; i < total_elem; i++){
        h_mem_ptr[i] = 1;
    }
    gpuDriverErrorCheck(hipHostGetDevicePointer(&h_mem, (void*)h_mem_ptr, 0));
    //gpuDriverErrorCheck(hipMemsetD32(h_mem,  (unsigned int)1, total_elem));

    hipDeviceptr_t d_mem;
    gpuDriverErrorCheck(hipMalloc(&d_mem, buf_size));
    gpuDriverErrorCheck(hipMemsetD32(d_mem,  (unsigned int)2, total_elem));

    hipDeviceptr_t h_tmp;
    int* h_tmp_ptr;
    gpuDriverErrorCheck(hipHostAlloc((void **)&h_tmp_ptr, buf_size,  hipHostMallocMapped)); 
    for(int i = 0; i < total_elem; i++){
        h_tmp_ptr[i] = 1;
    }
    gpuDriverErrorCheck(hipHostGetDevicePointer(&h_tmp, (void*)h_tmp_ptr, 0));
    //gpuDriverErrorCheck(hipMemsetD32(h_tmp,  (unsigned int)1, total_elem));

    hipDeviceptr_t d_tmp;
    gpuDriverErrorCheck(hipMalloc(&d_tmp, buf_size));
    gpuDriverErrorCheck(hipMemsetD32(d_tmp,  (unsigned int)3, total_elem));

    //Do memory copy and check result
    printf("Note: if no error message shows, then indicates test pass\n");
    gpuDriverErrorCheck(cuMemcpy(h_tmp, d_mem, buf_size));
    printf("Test memory copy D to H result....\n");
    checkDeviceResult<<<1,1>>>((int *)h_tmp, total_elem, 2);
    hipDeviceSynchronize();

    gpuDriverErrorCheck(cuMemcpy(h_tmp, h_mem, buf_size));
    printf("Test memory copy H to H result....\n");
    checkDeviceResult<<<1,1>>>((int *)h_tmp, total_elem, 1);
    //res = checkHostResult(h_tmp, total_elem, 1);
    hipDeviceSynchronize();
    
    gpuDriverErrorCheck(cuMemcpy(d_tmp, d_mem, buf_size));
    printf("Test memory copy D to D result....\n");
    checkDeviceResult<<<1,1>>>((int *)d_tmp, total_elem, 2);
    hipDeviceSynchronize();
    
    gpuDriverErrorCheck(cuMemcpy(d_tmp, h_mem, buf_size));
    printf("Test memory copy H to D result....\n");
    checkDeviceResult<<<1,1>>>((int *)d_tmp, total_elem, 1);
    hipDeviceSynchronize();

    return 0;
}